#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

const int BS = 16;  // Tamaño del bloque en la memoria compartida
// Kernel

__global__ void mikernel(float *A, float *B, float *C, int n){
    int tidx = (blockDim.x * blockIdx.x) + threadIdx.x;
    int tidy = (blockDim.y * blockIdx.y) + threadIdx.y;
    float acc = 0.0;
    for(int k=0; k < n; k++){
        acc += A[tidy * n + k] * B[k * n + tidx];
    }
    C[tidy * n + tidx] = acc;
}

__global__ void matrixMulShared(float *A, float *B, float *C, int n) {
    // Definir la memoria compartida
    __shared__ float sA[BS][BS];
    __shared__ float sB[BS][BS];

    int row = blockIdx.y * BS + threadIdx.y;
    int col = blockIdx.x * BS + threadIdx.x;

    float acc = 0.0;

    // Calcular el número de bloques necesarios
    int numBlocks = (n + BS - 1) / BS;

    for (int b = 0; b < numBlocks; ++b) {
        // Cargar bloques de A y B en memoria compartida
        if (row < n && (b * BS + threadIdx.x) < n) {
            sA[threadIdx.y][threadIdx.x] = A[row * n + b * BS + threadIdx.x];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }

        if (col < n && (b * BS + threadIdx.y) < n) {
            sB[threadIdx.y][threadIdx.x] = B[(b * BS + threadIdx.y) * n + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }

        // Sincronizar para asegurar que los bloques se han cargado completamente en memoria compartida
        __syncthreads();

        // Calcular producto parcial en el bloque
        for (int k = 0; k < BS; ++k) {
            acc += sA[threadIdx.y][k] * sB[k][threadIdx.x];
        }

        // Sincronizar antes de cargar el siguiente bloque
        __syncthreads();
    }

    // Escribir el resultado en la matriz de salida
    if (row < n && col < n) {
        C[row * n + col] = acc;
    }
}

void init_mat(float *M, int n, float c){
	long nelem = (long)n*(long)n;
	#pragma omp parallel for 
	//printf("nelem = %lu\n", nelem);
	for(long i=0; i<nelem; ++i){
		//printf("it %lu \n", i); fflush(stdout);
		M[i] = (((int)c*i) % 5)/100.0f;
	}
}

void print_mat(float *M, int n, const char *msg){
	if(n > 32){ return; }
	for(int i=0;i<n;i++){
		for(int j=0; j<n; j++){
            long index = i*n + j;
			//printf("%.3f ", M[index]);
		}
		//printf("\n");
	}
}

void cpu(float *A, float *B, float *C, int n){
	#pragma omp parallel for 
	for(int i=0;i<n;i++){
		for(int j=0; j<n; j++){
            long index = i*n + j;
			float acc = 0.0f;
			for(long k=0; k<n; k++){
				acc += A[(long)i*n + k]*B[k*n +j];	
			}
			C[index] = acc;
		}
	}
}

int main(int argc, char **argv) {
    if(argc != 5){
		fprintf(stderr, "error ejecutar como ./prog nt n mode BSIZE\n");
		exit(EXIT_FAILURE);
	}
    int nt, n, m, bs;
    float *A,  *B,  *C;
    float *dA, *dB, *dC;
    // obtener argumentos
    nt = atoi(argv[1]); // numero de threads (en caso del modo CPU)
    n = atoi(argv[2]);  // lado de una matrix de n x n
    m = atoi(argv[3]);  // modo (0 -> CPU   1 -> GPU)
    bs = atoi(argv[4]); // lado de un bloque de bs x bs

    long nelem = (long)n * (long)n;
    printf("nt = %i   n = %i  m = %i  bs = %i\n", nt, n, m, bs);
    printf("elementos matrix 	= %lu elementos\n", nelem);
    printf("memoria usada 	= %f GBs\n", sizeof(float)*nelem/1e9);
    omp_set_num_threads(nt);

    // inicializar arreglos en Host (CPU)
    double t1 = omp_get_wtime();
    printf("inicializando...."); fflush(stdout);
    A = new float[nelem]; B = new float[nelem]; C = new float[nelem];
    init_mat(A, n, 1); print_mat(A, n, "MATRIX A"); 
    init_mat(B, n, 2); print_mat(B, n, "MATRIX B");
    init_mat(C, n, 0);
    double t2 = omp_get_wtime();
    printf("ok: %f secs\n", t2-t1); fflush(stdout);

    // allocar memoria en device  (GPU)
    hipMalloc(&dA, sizeof(float) * nelem);
    hipMalloc(&dB, sizeof(float) * nelem);
    hipMalloc(&dC, sizeof(float) * nelem);

    // copiar de Host -> Device
    hipMemcpy(dA, A, sizeof(float)*nelem, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float)*nelem, hipMemcpyHostToDevice);

    dim3 block(bs, bs, 1);
    dim3 grid((n + bs -1) / bs, (n + bs -1) / bs, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    printf("calculando...."); fflush(stdout);
    hipEventRecord(start);
    if (m) {
        printf("GPU\n"); fflush(stdout);
        // Utilizar el nuevo kernel con memoria compartida
        matrixMulShared<<<grid, block>>>(dA, dB, dC, n);
    } else {
        printf("CPU\n"); fflush(stdout);
        cpu(A, B, C, n);
    }
    hipDeviceSynchronize(); hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    if (m) { hipMemcpy(C, dC, sizeof(float) * nelem, hipMemcpyDeviceToHost); }
    printf("ok: %f secs\n", milliseconds / 1000.0f); fflush(stdout);
    print_mat(C, n, "MATRIX C");
}
